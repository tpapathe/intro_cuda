
#include <hip/hip_runtime.h>
#include <stdio.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

// Size of arrays
#define N 1024

// Kernel
__global__ void dot_prod(int *a, int *b, int *res)
{
	__shared__ int products[N];

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	products[id] = a[id]*b[id];

	__syncthreads();

	if(id == 0)
	{
		int sum_of_products = 0;

		for(int i=0; i<N; i++)
		{
			sum_of_products = sum_of_products + products[i];
		}

		*res = sum_of_products;
	}
}

// Main program
int main()
{
	// Number of bytes to allocate for N integers
	size_t bytes = N*sizeof(int);

	// Allocate memory for arrays A, B, and result on host
	int *A = (int*)malloc(bytes);
	int *B = (int*)malloc(bytes);
	int *result = (int*)malloc(sizeof(int));

	// Allocate memory for arrays d_A, d_B, and d_result on device
	int *d_A, *d_B, *d_result;
	cudaErrorCheck( hipMalloc(&d_A, bytes) );
	cudaErrorCheck( hipMalloc(&d_B, bytes) );
	cudaErrorCheck( hipMalloc(&d_result, sizeof(int)) );

	// Fill host arrays A and B
  for(int i=0; i<N; i++)
  {
    A[i] = 1;
    B[i] = 2;
  }

  // Copy data from host arrays A and B to device arrays d_A and d_B
  cudaErrorCheck( hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice) );
  cudaErrorCheck( hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice) );

  // Set execution configuration parameters
  //    thr_per_blk: number of CUDA threads per grid block
  //    blk_in_grid: number of blocks in grid
  int thr_per_blk = 1024;
  int blk_in_grid = ceil( float(N) / thr_per_blk );

	// Launch kernel
	dot_prod<<< blk_in_grid, thr_per_blk >>>(d_A, d_B, d_result);

	  // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
  hipError_t cuErrSync  = hipGetLastError();

  // Check for errors on the GPU after control is returned to CPU
  hipError_t cuErrAsync = hipDeviceSynchronize();

  if (cuErrSync != hipSuccess) { printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrSync)); exit(0); }
  if (cuErrAsync != hipSuccess) { printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrAsync)); exit(0); }

	// Copy result from device to host
	cudaErrorCheck( hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost) );

	// Verify results
	if(*result != 2*N) { printf("Error: result is %d instead of %d\n", *result, 2*N); exit(0); }

	// Free CPU memory
	free(A);
	free(B);

	// Free GPU memory
	cudaErrorCheck( hipFree(d_A) );
	cudaErrorCheck( hipFree(d_B) );
	cudaErrorCheck( hipFree(d_result) );

  printf("\n---------------------------\n");
  printf("__SUCCESS__\n");
  printf("---------------------------\n");
  printf("N                 = %d\n", N);
  printf("Threads Per Block = %d\n", thr_per_blk);
  printf("Blocks In Grid    = %d\n", blk_in_grid);
  printf("---------------------------\n\n");
	
	return 0;
}
